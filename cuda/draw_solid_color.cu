#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "Shared.h"
#include <cuda/helpers.h>


extern "C" {
__constant__ Params params;
}

extern "C"
__global__ void __raygen__draw_solid_color()
{
    const uint3 launch_index = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    // Map our launch launch_index to a screen location and create a ray from 
    // the camera location through the screen
    float3 ray_origin, ray_direction;

    ray_origin.x = launch_index.x;
    ray_origin.y = launch_index.y;
    ray_origin.z = 0.f;

    // Normalized ray direction
    float sum = launch_index.x + launch_index.y + 1.f;
    ray_direction.x = launch_index.x / sum;
    ray_direction.y = launch_index.y / sum;
    ray_direction.z = 1.f / sum;
 
    // Trace the ray against our scene hierarchy
    unsigned int p0, p1, p2;
    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        0.0f,   // Min intersection distance
        1e16f,  // Max intersection distance
        0.0f,   // ray-time -- used for motion blur
        OptixVisibilityMask( 255 ), // Specify always visible
        OPTIX_RAY_FLAG_NONE,
        0,      // SBT offset -- See SBT discussion
        0,      // SBT stride -- See SBT discussion 
        0,      // missSBTIndex -- See SBT discussion
        p0, p1, p2 ); // These 32b values are the ray payload
 
    // Our results were packed into opaque 32b registers
    float3 result;
    result.x = int_as_float( p0 );
    result.y = int_as_float( p1 );
    result.z = int_as_float( p2 );
 
    // Record results in our output raster
    params.image[launch_index.y * params.image_width + launch_index.x] = make_color( result );
}
